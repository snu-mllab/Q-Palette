#include <cstdio>
#include <hip/hip_runtime.h>

#include "inference.h"
#include "inference.cu"

#define M 4096
#define N 1
#define K 4096

int main() {
    constexpr uint32_t S = 9;
    constexpr uint32_t R = 2;

    size_t m = M;
    size_t n = N;
    size_t k = K;

    float *out;
    uint32_t *compressed;
    half2 *x;
    half2 *codebook;
    uint32_t codebook_ones[(1 << R) * m];
    for (int i = 0; i < (1 << R) * m / 2; i++) {
        codebook_ones[i] = 0x3c003c00;
    }
    uint16_t x_ones[N * K];
    for (int i = 0; i < N * K; i++) {
        x_ones[i] = 0x3c00;
    }

    gpuErrchk(hipMalloc(&out, m * n * sizeof *out));
    gpuErrchk(hipMalloc(&compressed, m * k * R / CHAR_BIT));
    gpuErrchk(hipMalloc(&x, k * n * sizeof *x / 2));
    gpuErrchk(hipMalloc(&codebook, (1<<R) * m * sizeof(half)));

    gpuErrchk(hipMemset(out, 0, m * n * sizeof *out));
    gpuErrchk(hipMemset(compressed, 0, m * k * R / CHAR_BIT));
    gpuErrchk(hipMemcpy(x, x_ones, k * n * sizeof *x / 2, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(codebook, codebook_ones, (1<<R) * m * sizeof(half), hipMemcpyHostToDevice));

    decompress_matvec_ptr_sq<16, 9, R, 1, M, N, K, -1>((float*) out, (uint32_t*)compressed, (half2*)x, (half*)codebook, NULL);
    gpuErrchk(hipDeviceSynchronize());

    float *out_h;
    gpuErrchk(hipHostMalloc(&out_h, m * n * sizeof *out));
    gpuErrchk(hipMemcpy(out_h, out, m * n * sizeof *out, hipMemcpyDeviceToHost));
    int incorrect = 0;
    float sum = 0.f;
    for (uint32_t i = 0; i < m * n; i += 1) {
        incorrect += out_h[i] != K;
        sum += out_h[i];
        if (out_h[i] != K) {
            printf("incorrect: ref: %f actual: %f\n", (float)K, out_h[i]);
        }
    }
    printf("incorrect = %d\n", incorrect);
    printf("sum = %f\n", sum);
    gpuErrchk(hipHostFree(out_h));

    gpuErrchk(hipFree(out));
    gpuErrchk(hipFree(compressed));
    gpuErrchk(hipFree(x));
    gpuErrchk(hipFree(codebook));

    return incorrect;
}
